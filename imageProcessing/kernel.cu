#include "hip/hip_runtime.h"
#include <iostream>
#include "kernel.cuh"
#include "math_functions.h"
#include <vector>
using namespace std;

float elapsed;
hipEvent_t start, stop;
#define BLOCK_SIZE 32
// kernel function to gray an image
__global__ void grayscale(float *r, float *g, float *b, float *gray, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;	// width index
	int y = blockIdx.y * blockDim.y + threadIdx.y;	// height index

	if ((x < width) && (y < height)) {
		int idx = x * width + y;	// current pixel index

		//Gray = (Max(Red, Green, Blue) + Min(Red, Green, Blue)) / 2
		uint8_t calc = (fmaxf(r[idx], fmaxf(g[idx], b[idx])) + fminf(r[idx], fminf(g[idx], b[idx]))) / 2;

		gray[idx] = calc;
	}
}

// kernel function to blur a single color channel (R || G || B)
__global__ void convolute(float *ch, float *res, int height, int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;	// width index
	int y = blockIdx.y * blockDim.y + threadIdx.y;	// height index

	int radius = 8;
	float PI = atanf(1) * 4;
	if ((x < width) && (y < height)) {
		float sum = 0;
		float val = 0;
		int idx = x * width + y;	// current pixel index

		for (int i = y - radius; i < y + radius + 1; i++) {
			for (int j = x - radius; j < x + radius + 1; j++) {
				int h = fminf(height - 1, fmaxf(0, i));
				int w = fminf(width - 1, fmaxf(0, j));
				int dsq = (j - x) * (j - x) + (i - y) * (i - y);
				float wght = expf(-dsq / (2 * radius * radius)) / (PI * 2 * radius * radius);
				
				val += ch[w * width + h] * wght;
				sum += wght;
			}
		}
		res[idx] = round(val / sum);
	}
}

// deviceGrayscale allocates memory in host and device and copies data to and from host and device
// and also calls the appropriate kernel function for turning an image into black and white
vector<float> deviceGrayscale(float *r, float *g, float *b, int height, int width) {
	float *d_r;
	float *d_g;
	float *d_b;
	float *d_gray;

	float *h_gray;
	int size = height * width * sizeof(float);


	// start execution check
	startTime();

	h_gray = new float[size];
	checkCudaError(hipMalloc((void **)&d_r, size), "hipMalloc d_r");
	checkCudaError(hipMalloc((void **)&d_g, size), "hipMalloc d_g");
	checkCudaError(hipMalloc((void **)&d_b, size), "hipMalloc d_b");
	checkCudaError(hipMalloc((void **)&d_gray, size), "hipMalloc d_gray");

	checkCudaError(hipMemcpy(d_r, r, size, hipMemcpyHostToDevice), "hipMemcpy r to d_r");
	checkCudaError(hipMemcpy(d_g, g, size, hipMemcpyHostToDevice), "hipMemcpy g to d_g");
	checkCudaError(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice), "hipMemcpy b to d_b");

	int x = (int)(ceilf((float)(height) / BLOCK_SIZE));
	int y = (int)(ceilf((float)(width) / BLOCK_SIZE));

	const dim3 grid_size(x, y);
	const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

	grayscale <<<grid_size, block_size >>> (d_r, d_g, d_b, d_gray, height, width);

	checkCudaError(hipMemcpy(h_gray, d_gray, size, hipMemcpyDeviceToHost), "hipMemcpy d_gray to h_gray");

	checkCudaError(hipFree(d_r), "hipFree d_r");
	checkCudaError(hipFree(d_g), "hipFree d_g");
	checkCudaError(hipFree(d_b), "hipFree d_b");
	checkCudaError(hipFree(d_gray), "hipFree d_gray");

	// stop and print execution time
	stopTime("GRAYSCALE");

	vector<float> gray_channel;

	for (int i = 0; i < height * width; i++) {
		gray_channel.push_back(h_gray[i]);
	}

	delete[]h_gray;

	return gray_channel;
}

// deviceBlur allocates memory in host and device and copies data to and from host and device
// and also calls the appropriate kernel function to blur (gaussian) an image
tuple<vector<float>, vector<float>, vector<float>> deviceBlur(float *r, float *g, float *b, int height, int width) {
	float *d_r;
	float *d_g;
	float *d_b;
	float *d_blur;
	
	float *h_blur_r;
	float *h_blur_g;
	float *h_blur_b;

	int size = height * width * sizeof(float);

	h_blur_r = new float[size];
	h_blur_g = new float[size];
	h_blur_b = new float[size];


	checkCudaError(hipMalloc((void **)&d_r, size), "hipMalloc d_r");
	checkCudaError(hipMalloc((void **)&d_g, size), "hipMalloc d_g");
	checkCudaError(hipMalloc((void **)&d_b, size), "hipMalloc d_b");
	checkCudaError(hipMalloc((void **)&d_blur, size), "hipMalloc d_blur");

	startTime();

	checkCudaError(hipMemcpy(d_r, r, size, hipMemcpyHostToDevice), "hipMemcpy r to d_r");
	checkCudaError(hipMemcpy(d_g, g, size, hipMemcpyHostToDevice), "hipMemcpy g to d_g");
	checkCudaError(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice), "hipMemcpy b to d_b");

	int x = (int)(ceilf((float)(height) / BLOCK_SIZE));
	int y = (int)(ceilf((float)(width) / BLOCK_SIZE));

	const dim3 grid_size(x, y);
	const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);


	convolute << <grid_size, block_size >> > (d_r, d_blur, height, width);
	checkCudaError(hipMemcpy(h_blur_r, d_blur, size, hipMemcpyDeviceToHost), "hipMemcpy d_blur to h_blur_r");

	convolute << <grid_size, block_size >> > (d_g, d_blur, height, width);
	checkCudaError(hipMemcpy(h_blur_g, d_blur, size, hipMemcpyDeviceToHost), "hipMemcpy d_blur to h_blur_g");

	convolute << <grid_size, block_size >> > (d_b, d_blur, height, width);
	checkCudaError(hipMemcpy(h_blur_b, d_blur, size, hipMemcpyDeviceToHost), "hipMemcpy d_blur to h_blur_b");

	stopTime("BLUR");
	// free allocated memory on device
	checkCudaError(hipFree(d_r), "hipFree d_r");
	checkCudaError(hipFree(d_g), "hipFree d_g");
	checkCudaError(hipFree(d_b), "hipFree d_b");
	checkCudaError(hipFree(d_blur), "hipFree d_blur");

	

	vector<float> r_ch, g_ch, b_ch;

	for (int i = 0; i < height * width; i++) {
		r_ch.push_back(h_blur_r[i]);
		g_ch.push_back(h_blur_g[i]);
		b_ch.push_back(h_blur_b[i]);
	}

	// free allocated memory on host
	delete[]h_blur_r;
	delete[]h_blur_g;
	delete[]h_blur_b;
	
	
	return make_tuple(r_ch, g_ch, b_ch);
}

// Single stream (Stream 0)
tuple<vector<float>, vector<float>, vector<float>> deviceBlurSingleStream(float *r, float *g, float *b, int height, int width) {
	float *d_r;
	float *d_g;
	float *d_b;
	float *d_blur;

	float *h_blur_r;
	float *h_blur_g;
	float *h_blur_b;

	int size = height * width * sizeof(float);

	hipDeviceProp_t properties;
	int device;
	checkCudaError(hipGetDevice(&device), "hipGetDevice");
	checkCudaError(hipGetDeviceProperties(&properties, device), "hipGetDeviceProperties");
	// check if device handles overlaps
	if (!properties.deviceOverlap) {
		cout << "Device cannot handle overlaps\n";
		exit(1);
	}

	hipStream_t stream;
	checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");

	checkCudaError(hipHostMalloc((void **)&h_blur_r, size), "hipHostMalloc h_res_r");
	checkCudaError(hipHostMalloc((void **)&h_blur_g, size), "hipHostMalloc h_res_g");
	checkCudaError(hipHostMalloc((void **)&h_blur_b, size), "hipHostMalloc h_res_b");

	startTime();

	checkCudaError(hipMalloc((void **)&d_r, size), "hipMalloc d_r");
	checkCudaError(hipMalloc((void **)&d_g, size), "hipMalloc d_g");
	checkCudaError(hipMalloc((void **)&d_b, size), "hipMalloc d_b");
	checkCudaError(hipMalloc((void **)&d_blur, size), "hipMalloc d_blur");

	checkCudaError(hipMemcpyAsync(d_r, r, size, hipMemcpyHostToDevice, stream), "hipMemcpyAsync r to d_r");
	checkCudaError(hipMemcpyAsync(d_g, g, size, hipMemcpyHostToDevice, stream), "hipMemcpyAsync g to d_g");
	checkCudaError(hipMemcpyAsync(d_b, b, size, hipMemcpyHostToDevice, stream), "hipMemcpyAsync b to d_b");

	int x = (int)(ceilf((float)(height) / BLOCK_SIZE));
	int y = (int)(ceilf((float)(width) / BLOCK_SIZE));

	const dim3 grid_size(x, y);
	const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);


	convolute << <grid_size, block_size, 0, stream >> > (d_r, d_blur, height, width);
	checkCudaError(hipMemcpyAsync(h_blur_r, d_blur, size, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync d_blur to h_blur_r");

	convolute << <grid_size, block_size, 0, stream >> > (d_g, d_blur, height, width);
	checkCudaError(hipMemcpyAsync(h_blur_g, d_blur, size, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync d_blur to h_blur_g");

	checkCudaError(hipDeviceSynchronize(), "cudaThreadSync 1");

	convolute << <grid_size, block_size, 0, stream >> > (d_b, d_blur, height, width);
	checkCudaError(hipMemcpyAsync(h_blur_b, d_blur, size, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync d_blur to h_blur_b");

	checkCudaError(hipDeviceSynchronize(), "cudaThreadSync 2");

	checkCudaError(hipStreamSynchronize(stream), "cudaStreamnSync 1");

	// free allocated memory on device
	checkCudaError(hipFree(d_r), "hipFree d_r");
	checkCudaError(hipFree(d_g), "hipFree d_g");
	checkCudaError(hipFree(d_b), "hipFree d_b");
	checkCudaError(hipFree(d_blur), "hipFree d_blur");

	checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy");
	stopTime("BLUR - SINGLE STREAM");

	vector<float> r_ch, g_ch, b_ch;

	for (int i = 0; i < height * width; i++) {
		r_ch.push_back(h_blur_r[i]);
		g_ch.push_back(h_blur_g[i]);
		b_ch.push_back(h_blur_b[i]);
	}

	// free page-locked memory
	checkCudaError(hipHostFree(h_blur_r), "hipHostFree h_blur_r");
	checkCudaError(hipHostFree(h_blur_g), "hipHostFree h_blur_g");
	checkCudaError(hipHostFree(h_blur_b), "hipHostFree h_blur_b");


	return make_tuple(r_ch, g_ch, b_ch);
}

// Multiple streams
tuple<vector<float>, vector<float>, vector<float>> deviceBlurMultiStreams(float *r, float *g, float *b, int height, int width) {
	float *d_r;
	float *d_g;
	float *d_b;
	float *d_blur;

	float *h_blur_r;
	float *h_blur_g;
	float *h_blur_b;

	const int num_streams = 2;
	int size = height * width;
	int stream_size = size / num_streams;
	int stream_bytes = stream_size * sizeof(float);
	int bytes = size * sizeof(float);

	hipDeviceProp_t properties;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&properties, device);
	// check if device handles overlaps
	if (!properties.deviceOverlap) {
		cout << "Device cannot handle overlaps\n";
		exit(1);
	}

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) {
		checkCudaError(hipStreamCreate(&streams[i]), "stream create");
	}

	// allocate page-locked memory
	checkCudaError(hipHostMalloc((void **)&h_blur_r, bytes), "cuda malloc host h_blur_r");
	checkCudaError(hipHostMalloc((void **)&h_blur_g, bytes), "cuda malloc host h_blur_g");
	checkCudaError(hipHostMalloc((void **)&h_blur_b, bytes), "cuda malloc host h_blur_b");

	startTime();

	checkCudaError(hipMalloc((void **)&d_r, bytes), "cuda malloc dev d_r");
	checkCudaError(hipMalloc((void **)&d_g, bytes), "cuda malloc dev d_g");
	checkCudaError(hipMalloc((void **)&d_b, bytes), "cuda malloc dev d_b");
	checkCudaError(hipMalloc((void **)&d_blur, bytes), "cuda malloc dev d_blur");

	int x = (int)(ceilf((float)(height) / BLOCK_SIZE));
	int y = (int)(ceilf((float)(width) / BLOCK_SIZE));

	const dim3 grid_size(x, y);
	const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);

	for (int i = 0; i < num_streams; i++) {
		int offset = i * stream_size;

		checkCudaError(hipMemcpyAsync(&d_r[offset], &r[offset], stream_bytes, hipMemcpyHostToDevice, streams[i]), "cuda memcpy d_r");
		checkCudaError(hipMemcpyAsync(&d_g[offset], &g[offset], stream_bytes, hipMemcpyHostToDevice, streams[i]), "cuda memcpy d_g");
		checkCudaError(hipMemcpyAsync(&d_b[offset], &b[offset], stream_bytes, hipMemcpyHostToDevice, streams[i]), "cuda memcpy d_b");

		convolute << <grid_size, block_size, 0, streams[i] >> > (&d_r[offset], d_blur, height, width);
		checkCudaError(hipMemcpyAsync(&h_blur_r[offset], d_blur, stream_bytes, hipMemcpyDeviceToHost, streams[i]), "cuda memcpy h_blur_r");

		convolute << <grid_size, block_size, 0, streams[i] >> > (&d_g[offset], d_blur, height, width);
		checkCudaError(hipMemcpyAsync(&h_blur_g[offset], d_blur, stream_bytes, hipMemcpyDeviceToHost, streams[i]), "cuda memcpy h_blur_g");

		convolute << <grid_size, block_size, 0, streams[i] >> > (&d_b[offset], d_blur, height, width);
		checkCudaError(hipMemcpyAsync(&h_blur_b[offset], d_blur, stream_bytes, hipMemcpyDeviceToHost, streams[i]), "cuda memcpy h_blur_b");

		//checkCudaError(hipStreamSynchronize(streams[i]), "cuda stream sync");
	}

	checkCudaError(hipDeviceSynchronize(), "cudaDeviceSync");

	// free allocated memory on device
	checkCudaError(hipFree(d_r), "hipFree d_r");
	checkCudaError(hipFree(d_g), "hipFree d_g");
	checkCudaError(hipFree(d_b), "hipFree d_b");
	checkCudaError(hipFree(d_blur), "hipFree d_blur");

	for (int i = 0; i < num_streams; i++) {
		checkCudaError(hipStreamDestroy(streams[i]), "cudastreamdestroy");
	}

	stopTime("BLUR - MULTI STREAM");

	vector<float> r_ch, g_ch, b_ch;

	for (int i = 0; i < height * width; i++) {
		r_ch.push_back(h_blur_r[i]);
		g_ch.push_back(h_blur_g[i]);
		b_ch.push_back(h_blur_b[i]);
	}

	// free page-locked memory
	checkCudaError(hipHostFree(h_blur_r), "cudafreehost h_res_r");
	checkCudaError(hipHostFree(h_blur_g), "cudafreehost h_res_g");
	checkCudaError(hipHostFree(h_blur_b), "cudafreehost h_res_b");


	return make_tuple(r_ch, g_ch, b_ch);
}

// print error and exit if result is not hipSuccess
void checkCudaError(hipError_t result, char *loc) {
	if (result != hipSuccess) {
		cout << "ERROR at " << loc << " - " << hipGetErrorString(result) << endl;
		exit(1);
	}
}

void startTime() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

void stopTime(char *type) {
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed, start, stop);
	cout << "GPU Conversion Time (" << type << "): " << elapsed << " ms.\n";
}